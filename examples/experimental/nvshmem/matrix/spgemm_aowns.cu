#include "hip/hip_runtime.h"

#define __thrust_compiler_fence() __sync_synchronize()
#include <cusp/io/matrix_market.h>
#include <cusp/csr_matrix.h>
#include <cusp/array2d.h>
#include <cusp/multiply.h>
#include <cusp/array2d.h>
#include <cusp/print.h>

#include <bcl/bcl.hpp>
#include <bcl/backends/experimental/nvshmem/backend.hpp>
#include <bcl/containers/experimental/cuda/CudaMatrix.hpp>
#include <bcl/containers/experimental/cuda/launch_kernel.cuh>
#include <thrust/sort.h>

#include <bcl/containers/experimental/cuda/CudaSPMatrix.hpp>

#include <unordered_map>

#include "cusparse_util.hpp"
#include "spgemm.hpp"

#include <chrono>
#include <essl.h>

template <typename T, typename U>
struct PairHash {
  std::size_t operator()(const std::pair<T, U>& value) const noexcept {
    return std::hash<T>{}(value.first) ^ std::hash<U>{}(value.second);
  }
};

int main(int argc, char** argv) {
  BCL::init(16);
  BCL::cuda::init();

  using T = float;
  using index_type = int;

  bool verify_result = false;

  std::string fname = std::string(argv[1]);

  auto matrix_shape = BCL::matrix_io::matrix_info(fname);
  size_t m = matrix_shape.shape[0];
  size_t n = matrix_shape.shape[1];
  assert(m == n);
  size_t k = m;

  BCL::print("Choosing blocks...\n");
  auto blocks = BCL::block_matmul(m, n, k);

  BCL::print("Reading matrices...\n");
  BCL::cuda::SPMatrix<T, index_type> a(fname, std::move(blocks[0]));
  BCL::cuda::SPMatrix<T, index_type> b(fname, std::move(blocks[1]));
  BCL::cuda::SPMatrix<T, index_type> c(m, n, std::move(blocks[2]));

  BCL::print("Info:\n");
  if (BCL::rank() == 0) {
    printf("A:\n");
    a.print_info();
    printf("B:\n");
    b.print_info();
    printf("C:\n");
    c.print_info();
  }

  hipsparseStatus_t status = hipsparseCreate(&BCL::cuda::bcl_cusparse_handle_);
  BCL::cuda::throw_cusparse(status);
  status = hipsparseSetPointerMode(BCL::cuda::bcl_cusparse_handle_, HIPSPARSE_POINTER_MODE_HOST);
  BCL::cuda::throw_cusparse(status);

  // printf("A taking %lf GB, B %lf GB\n", 1.0e-9*a.my_mem(), 1.0e-9*b.my_mem());

  assert(a.grid_shape()[1] == b.grid_shape()[0]);

  using allocator_type = BCL::cuda::bcl_allocator<T>;

  BCL::cuda::duration_issue = 0;
  BCL::cuda::duration_sync = 0;
  BCL::cuda::duration_compute = 0;
  BCL::cuda::duration_accumulate = 0;
  BCL::cuda::duration_barrier = 0;

  BCL::print("Beginning SpGEMM...\n");

  BCL::barrier();
  auto begin = std::chrono::high_resolution_clock::now();
  BCL::cuda::gemm_aowns<T, index_type, allocator_type>(a, b, c);
  auto end = std::chrono::high_resolution_clock::now();
  double duration = std::chrono::duration<double>(end - begin).count();

  double max_issue = BCL::allreduce(BCL::cuda::duration_issue, BCL::max<double>{});
  double max_sync = BCL::allreduce(BCL::cuda::duration_sync, BCL::max<double>{});
  double max_compute = BCL::allreduce(BCL::cuda::duration_compute, BCL::max<double>{});
  double max_accumulate = BCL::allreduce(BCL::cuda::duration_accumulate, BCL::max<double>{});
  double max_barrier = BCL::allreduce(BCL::cuda::duration_barrier, BCL::max<double>{});

  double min_issue = BCL::allreduce(BCL::cuda::duration_issue, BCL::min<double>{});
  double min_sync = BCL::allreduce(BCL::cuda::duration_sync, BCL::min<double>{});
  double min_compute = BCL::allreduce(BCL::cuda::duration_compute, BCL::min<double>{});
  double min_accumulate = BCL::allreduce(BCL::cuda::duration_accumulate, BCL::min<double>{});
  double min_barrier = BCL::allreduce(BCL::cuda::duration_barrier, BCL::min<double>{});

  BCL::cuda::duration_issue = BCL::allreduce(BCL::cuda::duration_issue, std::plus<double>{});
  BCL::cuda::duration_sync = BCL::allreduce(BCL::cuda::duration_sync, std::plus<double>{});
  BCL::cuda::duration_compute = BCL::allreduce(BCL::cuda::duration_compute, std::plus<double>{});
  BCL::cuda::duration_accumulate = BCL::allreduce(BCL::cuda::duration_accumulate, std::plus<double>{});
  BCL::cuda::duration_barrier = BCL::allreduce(BCL::cuda::duration_barrier, std::plus<double>{});

  BCL::barrier();
  fflush(stdout);
  BCL::barrier();
  fprintf(stderr, "RANK(%lu) A has %lu nnz, B has %lu nnz, C has %lu nnz\n",
          BCL::rank(), a.my_nnzs(), b.my_nnzs(), c.my_nnzs());
  BCL::barrier();
  fflush(stderr);
  BCL::barrier();

  if (BCL::rank() == 0) {
    printf("duration_issue %lf (%lf -> %lf)\n",
           BCL::cuda::duration_issue / BCL::nprocs(),
           min_issue, max_issue);
    printf("duration_sync %lf (%lf -> %lf)\n",
           BCL::cuda::duration_sync / BCL::nprocs(),
           min_sync, max_sync);
    printf("duration_compute %lf (%lf -> %lf)\n",
           BCL::cuda::duration_compute / BCL::nprocs(),
           min_compute, max_compute);
    printf("duration_accumulate %lf (%lf -> %lf)\n",
           BCL::cuda::duration_accumulate / BCL::nprocs(),
           min_accumulate, max_accumulate);
    printf("duration_barrier %lf (%lf -> %lf)\n",
           BCL::cuda::duration_barrier / BCL::nprocs(),
           min_barrier, max_barrier);
  }

  BCL::barrier();
  fflush(stdout);
  BCL::barrier();

  BCL::print("Matrix multiply finished in %lf s\n", duration);

  if (BCL::rank() == 0 && verify_result) {
    BCL::CSRMatrix<T, index_type> mat(fname);

    auto local_a = BCL::cuda::to_gpu<T, index_type, allocator_type>(mat);

    auto s_c = spgemm_cusparse(local_a, local_a);

    fprintf(stderr, "Getting COO...\n");
    auto local_c = c.get().get_coo();
    local_c = BCL::cuda::remove_zeros(local_c);

    auto s_c_coo = BCL::cuda::to_cpu(s_c).get_coo();

    fprintf(stderr, "local_computation (%lu nnz), distributed result (%lu nnz)\n", s_c_coo.size(), local_c.size());

/*
    FILE* f = fopen("/gpfs/alpine/bif115/scratch/b2v/data/distributed.binary", "w");
    assert(f != NULL);
    using tuple_type = typename decltype(local_c)::value_type;
    fwrite(&local_c[0], sizeof(tuple_type), local_c.size(), f);
    fclose(f);

    f = fopen("/gpfs/alpine/bif115/scratch/b2v/data/sequential.binary", "w");
    assert(f != NULL);
    using tuple_type = typename decltype(local_c)::value_type;
    fwrite(&s_c_coo[0], sizeof(tuple_type), s_c_coo.size(), f);
    fclose(f);
    */

    if (s_c_coo.size() != local_c.size()) {
      fprintf(stderr, "ERROR: number of nonzeros does not match.\n");
    } else {
      fprintf(stderr, "Nonzeros match %lu == %lu\n", s_c_coo.size(), local_c.size());
    }

/*
    using coord_type = std::pair<index_type, index_type>;
    std::unordered_map<coord_type, T, PairHash<index_type, index_type>> serial_set;
    std::unordered_map<coord_type, T, PairHash<index_type, index_type>> distr_set;

    fprintf(stderr, "Building serial set.\n");
    auto begin = std::chrono::high_resolution_clock::now();
    for (const auto& nz : s_c_coo) {
      auto idx = std::get<0>(nz);
      auto val = std::get<1>(nz);
      serial_set[idx] = val;
    }
    auto end = std::chrono::high_resolution_clock::now();
    double duration = std::chrono::duration<double>(end - begin).count();
    fprintf(stderr, "Took %lf s\n", duration);

    fprintf(stderr, "Building distributed set.\n");
    begin = std::chrono::high_resolution_clock::now();
    for (const auto& nz : local_c) {
      auto idx = std::get<0>(nz);
      auto val = std::get<1>(nz);
      distr_set[idx] = val;
    }
    end = std::chrono::high_resolution_clock::now();
    duration = std::chrono::duration<double>(end - begin).count();
    fprintf(stderr, "Took %lf s\n", duration);

    fprintf(stderr, "Looking through serial set to see if distributed set matches.\n");
    T eps = 1.0e-5;
    for (const auto& nz : s_c_coo) {
      auto idx = std::get<0>(nz);
      auto val = std::get<1>(nz);

      if (distr_set.find(idx) == distr_set.end()) {
        fprintf(stderr, "Serial result contains (%lu, %lu, %f) not in distributed result.\n",
                idx.first, idx.second, val);
      } else if (std::abs(val - distr_set[idx]) > eps) {
        fprintf(stderr, "Distributed result (%lu, %lu, %f) != serial result (%lu, %lu, %f)\n",
                idx.first, idx.second, distr_set[idx],
                idx.first, idx.second, val);
      }
    }

    fprintf(stderr, "Looking through distributed set to see if distributed set matches.\n");
    for (const auto& nz : local_c) {
      auto idx = std::get<0>(nz);
      auto val = std::get<1>(nz);

      if (serial_set.find(idx) == serial_set.end()) {
        fprintf(stderr, "Distributed result contains (%lu, %lu, %f) not in serial result.\n",
                idx.first, idx.second, val);
      }
    }

*/
    T eps = 1.0e-5;
    for (size_t i = 0; i < s_c_coo.size(); i++) {
      auto idx_a = std::get<0>(s_c_coo[i]);
      auto idx_b = std::get<0>(local_c[i]);

      auto val_a = std::get<1>(s_c_coo[i]);
      auto val_b = std::get<1>(local_c[i]);

      assert(idx_a == idx_b);
      if (std::abs((val_a - val_b)/val_b) >= eps) {
        fprintf(stderr, "(%lu, %lu) == (%lu, %lu)\n", idx_a.first, idx_a.second,
                                                      idx_b.first, idx_b.second);
        fprintf(stderr, "%f ~= %f\n", val_a, val_b);
        fflush(stderr);
      }
      assert(std::abs((val_a - val_b)/val_b) < eps);
      // printf("(%lu, %lu) == (%lu, %lu)\n", idx_a.first, idx_a.second,
      //                                    idx_b.first, idx_b.second);
      // printf("%f ~= %f\n", val_a, val_b);
    }
    printf("OK!\n");
  }

  BCL::finalize();
  return 0;
}
