// SPDX-FileCopyrightText: 2021 Benjamin Brock
//
// SPDX-License-Identifier: BSD-3-Clause

#include <bcl/bcl.hpp>
#include <bcl/backends/experimental/nvshmem/backend.hpp>
#include <bcl/containers/experimental/cuda/DuplQueue.hpp>
#include <bcl/containers/experimental/cuda/launch_kernel.cuh>

#include <chrono>

#define NUM_INSERTS 2*1024

int main(int argc, char** argv) {
  BCL::init(16);

  printf("Hello, world! I am rank %lu/%lu\n",
         BCL::rank(), BCL::nprocs());

  BCL::cuda::init(16*1024);

  size_t num_inserts = NUM_INSERTS;
  size_t insert_size = 64;

  BCL::cuda::DuplQueue<int> queue(0, num_inserts*insert_size);

  BCL::cuda::device_vector<int> values(insert_size);
  std::vector<int> values_local(insert_size, BCL::rank());
  values.assign(values_local.begin(), values_local.end());

  BCL::cuda::barrier();
  auto begin = std::chrono::high_resolution_clock::now();

  BCL::cuda::global_launch(num_inserts,
                     [] __device__ (size_t idx, BCL::cuda::DuplQueue<int>& queue,
                                    BCL::cuda::device_vector<int>& values) {
                       bool success = queue.push(values.data(), values.size());
                       if (!success) {
                         printf("AGH! I have failed!\n");
                       }
                     }, queue, values);

  hipDeviceSynchronize();

  fflush(stdout);
  fflush(stderr);
  BCL::barrier();
  fflush(stdout);
  fflush(stderr);
  BCL::barrier();

  BCL::print("Here...\n");

  BCL::cuda::barrier();
  BCL::print("After barrier...\n");
  auto end = std::chrono::high_resolution_clock::now();

  BCL::finalize();
  return 0;
}
