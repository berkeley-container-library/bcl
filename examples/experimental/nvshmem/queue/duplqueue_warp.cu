// SPDX-FileCopyrightText: 2021 Benjamin Brock
//
// SPDX-License-Identifier: BSD-3-Clause

#include <bcl/bcl.hpp>
#include <bcl/backends/experimental/nvshmem/backend.hpp>
#include <bcl/containers/experimental/cuda/DuplQueue.hpp>
#include <bcl/containers/experimental/cuda/launch_kernel.cuh>

#include <chrono>

#define NUM_INSERTS 2*8*1024

int main(int argc, char** argv) {
  BCL::init(16);

  printf("Hello, world! I am rank %lu/%lu\n",
         BCL::rank(), BCL::nprocs());

  BCL::cuda::init(14*1024);

  size_t num_inserts = NUM_INSERTS;
  size_t insert_size = 8*1024;

  // Round up so each rank has an equal number of inserts.
  size_t inserts_per_rank = (num_inserts + BCL::nprocs() - 1) / BCL::nprocs();
  inserts_per_rank *= BCL::nprocs();
  num_inserts = inserts_per_rank * BCL::nprocs();

  BCL::cuda::DuplQueue<int> queue(0, num_inserts*insert_size);

  BCL::cuda::device_vector<int, BCL::cuda::bcl_allocator<int>> values(insert_size);
  // BCL::cuda::device_vector<int> values(insert_size);
  std::vector<int> values_local(insert_size, BCL::rank());
  values.assign(values_local.begin(), values_local.end());

  BCL::cuda::barrier();
  auto begin = std::chrono::high_resolution_clock::now();

  BCL::cuda::launch(inserts_per_rank*32,
                    [] __device__ (size_t idx, BCL::cuda::DuplQueue<int>& queue,
                                   BCL::cuda::device_vector<int, BCL::cuda::bcl_allocator<int>>& values) {
                      // BCL::cuda::device_vector<int>& values) {
                      bool success = queue.push_warp(values.data(), values.size());
                      if (!success) {
                        printf("AGH! I have failed!\n");
                      }
                    }, queue, values);

  hipDeviceSynchronize();
  BCL::cuda::barrier();
  auto end = std::chrono::high_resolution_clock::now();

  double duration = std::chrono::duration<double>(end - begin).count();

  double data_moved = num_inserts*insert_size*sizeof(int);
  double data_moved_gb = data_moved*1e-9;

  double bw = data_moved / duration;
  double bw_gb = bw*1e-9;

  BCL::print("Total %lf s (%lf GB) (%lf GB/s)\n", duration, data_moved_gb, bw_gb);

  if (BCL::rank() == 0) {
    BCL::cuda::launch(num_inserts,
                      [] __device__ (size_t idx, BCL::cuda::DuplQueue<int>& queue) {
                        int value = 12;
                        bool success = queue.local_pop(value);
                        // printf("%lu: %d (%s)\n", idx, value, (success) ? "success" : "failure");
                      }, queue);
    hipDeviceSynchronize();
  }
  BCL::cuda::barrier();

  BCL::print("Here...\n");

  BCL::cuda::barrier();
  BCL::print("After barrier...\n");

  BCL::finalize();
  return 0;
}
